#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    float *hx, *hy, *dx, *dy;
    hx = (float*)malloc(N * sizeof(float));
    hy = (float*)malloc(N * sizeof(float));

    hipMalloc(&dx, N * sizeof(float));
    hipMalloc(&dy, N * sizeof(float));

    for (int idx = 0; idx < N; idx++)
    {
        hx[idx] = 1.0f;
        hy[idx] = 2.0f;
    }

    hipMemcpy(dx, hx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, N * sizeof(float), hipMemcpyHostToDevice);

    unsigned int threads = 256;
    unsigned int blocks = (N + 255) / threads;

    float ratio = 2.0f;

    //saxpy<<<blocks, threads>>>(N, ratio, dx, dy);

    void *args[] = { &N, &ratio, &dx, &dy };
    hipLaunchKernel((void*)saxpy, dim3(blocks), dim3(threads), args, 0, NULL);

    hipMemcpy(hy, dy, N * sizeof(float), hipMemcpyDeviceToHost);

    float max_error = 0.0f;
    for (int jdx = 0; jdx < N; jdx++)
    {
        max_error = max(max_error, abs(hy[jdx] - 4.0f));
    }

    printf("Max Error: %f\n", max_error);

    hipFree(dx);
    hipFree(dy);
    free(hx);
    free(hy);

    return 0;
}
