#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__
void dummy0()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d in block %d\n", i, blockIdx.x);
}


int main(void)
{
    int N=1 << 20;
    unsigned int threads = 256;
    unsigned int blocks = (N + 255) / threads;

//    dummy0<<<blocks, threads>>>();


    void *args[] = {};

    printf("hipLaunchKernel with 0 arguments\n");
    hipError_t hipError_t = hipLaunchKernel(reinterpret_cast<const void*>((void*)dummy0), dim3(blocks), dim3(threads), args, 0, NULL);

    printf("cudaError:%d\n",hipError_t);
    return 0;
}
