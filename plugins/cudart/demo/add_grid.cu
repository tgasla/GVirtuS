
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20;
    float *x, *y;

    printf("Allocate Unified Memory -- accessible from CPU or GPU\n");
    gpuErrchk(hipMallocManaged(&x, N*sizeof(float)));
    gpuErrchk(hipMallocManaged(&y, N*sizeof(float)));

    printf("Initialize x and y arrays on the host\n");
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    printf("Launch kernel on 1M elements on the GPU\n");
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    printf("Wait for GPU to finish before accessing on host\n");
    gpuErrchk(hipDeviceSynchronize());

    printf("Check for errors (all values should be 3.0f)\n");
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    printf("Free memory\n");
    gpuErrchk(hipFree(x));
    gpuErrchk(hipFree(y));

    return 0;
}