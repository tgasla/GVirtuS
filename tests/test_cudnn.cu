#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA Error: " << hipGetErrorString(err)
#define CUDNN_CHECK(err) ASSERT_EQ((err), HIPDNN_STATUS_SUCCESS) << "CUDNN Error: " << hipdnnGetErrorString(err)

TEST(cuDNN, CreateDestroy) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, GetVersion) {
    size_t version = hipdnnGetVersion();
    ASSERT_GT(version, 0);
    // std::cout << "cuDNN version: " << version << std::endl;
}

TEST(cuDNN, SetGetStream) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDNN_CHECK(hipdnnSetStream(handle, stream));

    hipStream_t got_stream;
    CUDNN_CHECK(hipdnnGetStream(handle, &got_stream));
    ASSERT_EQ(stream, got_stream);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, TensorDescriptorCreateDestroy) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}

TEST(cuDNN, TensorDescriptorSetGet) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 2, 3, 4));

    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;

    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(desc, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride));
    ASSERT_EQ(n, 1);
    ASSERT_EQ(c, 2);
    ASSERT_EQ(h, 3);
    ASSERT_EQ(w, 4);

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}

TEST(cuDNN, TensorNdDescriptorSetGet) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));

    const int nbDims = 4;
    int dimA[nbDims]    = {1, 2, 3, 4};  // NCHW
    int strideA[nbDims] = {
        2 * 3 * 4,  // N stride = C * H * W
        3 * 4,      // C stride = H * W
        4,          // H stride = W
        1           // W stride = 1
    };

    CUDNN_CHECK(hipdnnSetTensorNdDescriptor(desc, HIPDNN_DATA_FLOAT, nbDims, dimA, strideA));

    // Prepare to retrieve
    hipdnnDataType_t dataTypeOut;
    int nbDimsOut;
    int dimOut[nbDims];
    int strideOut[nbDims];

    CUDNN_CHECK(hipdnnGetTensorNdDescriptor(desc, nbDims, &dataTypeOut, &nbDimsOut, dimOut, strideOut));
    ASSERT_EQ(dataTypeOut, HIPDNN_DATA_FLOAT);
    ASSERT_EQ(nbDimsOut, nbDims);
    for (int i = 0; i < nbDims; ++i) {
        ASSERT_EQ(dimOut[i], dimA[i]);
        ASSERT_EQ(strideOut[i], strideA[i]);
    }

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}

TEST(cuDNN, ActivationForwardReLU) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));

    hipdnnActivationDescriptor_t actDesc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

    float h_input[] = {-1.0f, 0.0f, 2.0f};
    float h_output[3] = {};

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(h_input)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(h_output)));
    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnActivationForward(handle, actDesc, &alpha, desc, d_input, &beta, desc, d_output));
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    ASSERT_FLOAT_EQ(h_output[0], 0.0f);
    ASSERT_FLOAT_EQ(h_output[1], 0.0f);
    ASSERT_FLOAT_EQ(h_output[2], 2.0f);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(actDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, PoolingForwardMax) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t in_desc, out_desc;
    hipdnnPoolingDescriptor_t pool_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pool_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 2, 2));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            2, 2, 0, 0, 1, 1));

    float h_input[] = {1, 2, 3, 4}; // max is 4
    float h_output[1] = {};

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(h_input)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(h_output)));
    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnPoolingForward(handle, pool_desc, &alpha, in_desc, d_input, &beta, out_desc, d_output));
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    ASSERT_FLOAT_EQ(h_output[0], 4.0f);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(pool_desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, FilterDescriptorCreateDestroy) {
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
}

TEST(cuDNN, ErrorString) {
    const char* msg = hipdnnGetErrorString(HIPDNN_STATUS_ALLOC_FAILED);
    ASSERT_TRUE(msg != nullptr);
}

TEST(cuDNN, ConvolutionForward) {
    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    // Input dimensions: NCHW
    int n = 1, c = 1, h = 5, w = 5;
    int kernel_size = 3;
    int pad = 0, stride = 1, dilation = 1;

    // Output dimensions
    int out_n = 1, out_c = 1;
    int out_h = (h + 2 * pad - dilation * (kernel_size - 1) - 1) / stride + 1;
    int out_w = (w + 2 * pad - dilation * (kernel_size - 1) - 1) / stride + 1;

    // Allocate and initialize host memory
    std::vector<float> h_input(n * c * h * w, 1.0f);   // Input filled with ones
    std::vector<float> h_filter(out_c * c * kernel_size * kernel_size, 1.0f); // Filter filled with ones
    std::vector<float> h_output(out_n * out_c * out_h * out_w, 0.0f); // Output initialized to zeros

    // Allocate device memory
    float *d_input, *d_filter, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, h_input.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_filter, h_filter.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, h_output.size() * sizeof(float)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter, h_filter.data(), h_filter.size() * sizeof(float), hipMemcpyHostToDevice));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, out_c, c, kernel_size, kernel_size));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, pad, pad, stride, stride, dilation, dilation, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Set output descriptor
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

    // Select convolution algorithm
    hipdnnConvolutionFwdAlgoPerf_t perf_results;
    int returned_algo_count = 0;
    CUDNN_CHECK(hipdnnFindConvolutionForwardAlgorithm(
        cudnn, input_desc, filter_desc, conv_desc, output_desc,
        1, &returned_algo_count, &perf_results));
    hipdnnConvolutionFwdAlgo_t algo = perf_results.algo;

    // Allocate workspace
    size_t workspace_bytes = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, filter_desc, conv_desc, output_desc,
                                                        algo, &workspace_bytes));
    void* d_workspace = nullptr;
    if (workspace_bytes > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspace_bytes));
    }

    // Perform the convolution
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha, input_desc, d_input, filter_desc, d_filter,
                                        conv_desc, algo, d_workspace, workspace_bytes, &beta, output_desc, d_output));

    // Copy the result back to host
    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, h_output.size() * sizeof(float), hipMemcpyDeviceToHost));

    // Verify the result
    for (size_t i = 0; i < h_output.size(); ++i) {
       ASSERT_NEAR(h_output[i], 9.0f, 1e-5f) << "Mismatch at index " << i;
    }

    // Cleanup
    if (d_workspace) CUDA_CHECK(hipFree(d_workspace));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, ConvolutionBackwardData) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    // Dimensions
    int n = 1, c = 1, h = 5, w = 5;
    int kernel_size = 3;
    int pad = 0, stride = 1, dilation = 1;
    int out_h = (h + 2 * pad - dilation * (kernel_size - 1) - 1) / stride + 1;
    int out_w = (w + 2 * pad - dilation * (kernel_size - 1) - 1) / stride + 1;

    // Host-side buffers
    std::vector<float> h_input(n * c * h * w, 1.0f); // input filled with ones
    std::vector<float> h_filter(c * kernel_size * kernel_size, 1.0f); // filter filled with ones
    std::vector<float> h_output_grad(n * c * out_h * out_w, 1.0f); // output gradient (dy) filled with ones
    std::vector<float> h_input_grad(n * c * h * w, 0.0f); // dx to be filled

    // Device-side buffers
    float *d_input, *d_filter, *d_output_grad, *d_input_grad;
    CUDA_CHECK(hipMalloc(&d_input, h_input.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_filter, h_filter.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output_grad, h_output_grad.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_input_grad, h_input_grad.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_filter, h_filter.data(), h_filter.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_output_grad, h_output_grad.data(), h_output_grad.size() * sizeof(float), hipMemcpyHostToDevice));

    // Descriptors
    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, out_h, out_w));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, c, c, kernel_size, kernel_size));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, pad, pad, stride, stride, dilation, dilation,
                                                HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Workspace
    size_t workspace_bytes = 0;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        cudnn, filter_desc, output_desc, conv_desc, input_desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, &workspace_bytes));

    void* d_workspace = nullptr;
    if (workspace_bytes > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspace_bytes));
    }

    // Compute backward data (dx)
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionBackwardData(
        cudnn, &alpha, filter_desc, d_filter, output_desc, d_output_grad,
        conv_desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, d_workspace, workspace_bytes,
        &beta, input_desc, d_input_grad));

    // Copy result back
    CUDA_CHECK(hipMemcpy(h_input_grad.data(), d_input_grad, h_input_grad.size() * sizeof(float), hipMemcpyDeviceToHost));

    // Verify (each element in dx should be equal to number of times the kernel overlaps it)
    // With 3x3 kernel and 5x5 input, interior pixels get full 3x3 overlap = 9
    for (int i = 0; i < h; ++i) {
        for (int j = 0; j < w; ++j) {
            int count = 0;
            for (int m = 0; m < kernel_size; ++m) {
                for (int n = 0; n < kernel_size; ++n) {
                    int y = i - m + pad;
                    int x = j - n + pad;
                    if (y >= 0 && y < out_h && x >= 0 && x < out_w)
                        count++;
                }
            }
            float expected = static_cast<float>(count);  // number of times the kernel covered this pixel
            int idx = i * w + j;
            ASSERT_NEAR(h_input_grad[idx], expected, 1e-5f) << "Mismatch at (" << i << "," << j << ")";
        }
    }

    // Cleanup
    if (d_workspace) CUDA_CHECK(hipFree(d_workspace));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_output_grad));
    CUDA_CHECK(hipFree(d_input_grad));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, ConvolutionBackwardFilter) {
    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    int n = 1, c = 1, h = 5, w = 5;
    int kernel_size = 3;
    int pad = 0, stride = 1, dilation = 1;

    int out_h = (h + 2 * pad - dilation * (kernel_size - 1) - 1) / stride + 1;
    int out_w = out_h;
    int out_n = n, out_c = c;

    std::vector<float> h_input(n * c * h * w, 1.0f);            // Input filled with 1.0
    std::vector<float> h_output_grad(n * c * out_h * out_w, 1.0f); // dOutput filled with 1.0
    std::vector<float> h_filter_grad(c * c * kernel_size * kernel_size, 0.0f); // dFilter initialized to 0.0

    float *d_input, *d_output_grad, *d_filter_grad;
    CUDA_CHECK(hipMalloc(&d_input, h_input.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output_grad, h_output_grad.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_filter_grad, h_filter_grad.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_output_grad, h_output_grad.data(), h_output_grad.size() * sizeof(float), hipMemcpyHostToDevice));

    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, out_c, c, kernel_size, kernel_size));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, pad, pad, stride, stride, dilation, dilation,
                                                HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Get algorithm using the v7 API
    int returned_algo_count = 0;
    hipdnnConvolutionBwdFilterAlgoPerf_t perf_results;

    CUDNN_CHECK(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
        cudnn,
        input_desc,
        output_desc,
        conv_desc,
        filter_desc,
        1,
        &returned_algo_count,
        &perf_results));

    ASSERT_GT(returned_algo_count, 0);
    hipdnnConvolutionBwdFilterAlgo_t algo = perf_results.algo;

    // Allocate workspace
    size_t workspace_bytes = 0;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        cudnn, input_desc, output_desc, conv_desc, filter_desc, algo, &workspace_bytes));

    void* d_workspace = nullptr;
    if (workspace_bytes > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspace_bytes));
    }

    // Run convolution backward filter
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
        cudnn, &alpha, input_desc, d_input, output_desc, d_output_grad, conv_desc, algo,
        d_workspace, workspace_bytes, &beta, filter_desc, d_filter_grad));

    CUDA_CHECK(hipMemcpy(h_filter_grad.data(), d_filter_grad,
                          h_filter_grad.size() * sizeof(float), hipMemcpyDeviceToHost));

    // Verify: each value in the filter gradient should be equal to 9.0 (from 3x3 patch of 1s)
    for (size_t i = 0; i < h_filter_grad.size(); ++i) {
        ASSERT_NEAR(h_filter_grad[i], 9.0f, 1e-5f) << "Mismatch at index " << i;
    }

    // Cleanup
    if (d_workspace) CUDA_CHECK(hipFree(d_workspace));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output_grad));
    CUDA_CHECK(hipFree(d_filter_grad));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, ActivationBackwardReLU) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 2, w = 3;
    const int total_size = n * c * h * w;

    std::vector<float> h_input = {
        -1.0f, 0.0f, 1.0f,
         2.0f, -3.0f, 4.0f
    }; // Input before ReLU
    std::vector<float> h_output = {
         0.0f, 0.0f, 1.0f,
         2.0f, 0.0f, 4.0f
    }; // Output after ReLU
    std::vector<float> h_grad_output(total_size, 1.0f); // dY (incoming gradient): all ones
    std::vector<float> h_grad_input(total_size, 0.0f);  // dX: what we compute

    float *d_input, *d_output, *d_grad_output, *d_grad_input;
    CUDA_CHECK(hipMalloc(&d_input, total_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, total_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad_output, total_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad_input, total_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), total_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_output, h_output.data(), total_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_grad_output, h_grad_output.data(), total_size * sizeof(float), hipMemcpyHostToDevice));

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

    hipdnnActivationDescriptor_t act_desc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act_desc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_RELU,
                                             HIPDNN_PROPAGATE_NAN, 0.0));

    float alpha = 1.0f, beta = 0.0f;

    CUDNN_CHECK(hipdnnActivationBackward(
        cudnn, act_desc,
        &alpha,
        desc, d_output,        // y
        desc, d_grad_output,   // dy
        desc, d_input,         // x
        &beta,
        desc, d_grad_input     // dx
    ));

    CUDA_CHECK(hipMemcpy(h_grad_input.data(), d_grad_input,
                          total_size * sizeof(float), hipMemcpyDeviceToHost));

    std::vector<float> expected_grad_input = {
        0.0f, 0.0f, 1.0f,
        1.0f, 0.0f, 1.0f
    };

    for (int i = 0; i < total_size; ++i) {
        ASSERT_NEAR(h_grad_input[i], expected_grad_input[i], 1e-5f) << "Mismatch at index " << i;
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_grad_output));
    CUDA_CHECK(hipFree(d_grad_input));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(act_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, BatchNormForwardTraining) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 2, h = 1, w = 3; // Small 1x2x1x3 tensor
    const int total_size = n * c * h * w;
    const int param_size = c;

    std::vector<float> h_input = {
        1.0f, 2.0f, 3.0f,   // Channel 0
        4.0f, 5.0f, 6.0f    // Channel 1
    };
    std::vector<float> h_grad_output(total_size, 1.0f);  // dy
    std::vector<float> h_output(total_size, 0.0f);
    std::vector<float> h_grad_input(total_size, 0.0f);

    std::vector<float> h_scale(param_size, 1.0f);  // gamma
    std::vector<float> h_bias(param_size, 0.0f);   // beta
    std::vector<float> h_running_mean(param_size, 0.0f);
    std::vector<float> h_running_var(param_size, 1.0f);

    std::vector<float> h_saved_mean(param_size);
    std::vector<float> h_saved_var(param_size);

    std::vector<float> h_dscale(param_size, 0.0f);
    std::vector<float> h_dbias(param_size, 0.0f);

    float *d_input, *d_output, *d_grad_output, *d_grad_input;
    float *d_scale, *d_bias, *d_running_mean, *d_running_var;
    float *d_saved_mean, *d_saved_var;
    float *d_dscale, *d_dbias;

    CUDA_CHECK(hipMalloc(&d_input, total_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, total_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad_output, total_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad_input, total_size * sizeof(float)));

    CUDA_CHECK(hipMalloc(&d_scale, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_bias, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_running_mean, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_running_var, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_saved_mean, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_saved_var, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dscale, param_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dbias, param_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), total_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_grad_output, h_grad_output.data(), total_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_scale, h_scale.data(), param_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bias, h_bias.data(), param_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_running_mean, h_running_mean.data(), param_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_running_var, h_running_var.data(), param_size * sizeof(float), hipMemcpyHostToDevice));

    hipdnnTensorDescriptor_t x_desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&x_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

    hipdnnTensorDescriptor_t bn_desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&bn_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(bn_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, c, 1, 1));

    float alpha = 1.0f, beta = 0.0f;
    double epsilon = 1e-5;
    double exponential_average_factor = 1.0;

    // Forward training pass
    CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
        cudnn,
        HIPDNN_BATCHNORM_SPATIAL,
        &alpha, &beta,
        x_desc, d_input,
        x_desc, d_output,
        bn_desc, d_scale, d_bias,
        exponential_average_factor,
        d_running_mean, d_running_var,
        epsilon,
        d_saved_mean, d_saved_var
    ));

    // Cleanup
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_grad_output));
    CUDA_CHECK(hipFree(d_grad_input));
    CUDA_CHECK(hipFree(d_scale));
    CUDA_CHECK(hipFree(d_bias));
    CUDA_CHECK(hipFree(d_running_mean));
    CUDA_CHECK(hipFree(d_running_var));
    CUDA_CHECK(hipFree(d_saved_mean));
    CUDA_CHECK(hipFree(d_saved_var));
    CUDA_CHECK(hipFree(d_dscale));
    CUDA_CHECK(hipFree(d_dbias));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(x_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bn_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

// TEST(cuDNN, BatchNormForwardTrainingAndBackward) {
//     hipdnnHandle_t cudnn;
//     CUDNN_CHECK(hipdnnCreate(&cudnn));

//     const int n = 1, c = 2, h = 1, w = 3; // Small 1x2x1x3 tensor
//     const int total_size = n * c * h * w;
//     const int param_size = c;

//     std::vector<float> h_input = {
//         1.0f, 2.0f, 3.0f,   // Channel 0
//         4.0f, 5.0f, 6.0f    // Channel 1
//     };
//     std::vector<float> h_grad_output(total_size, 1.0f);  // dy
//     std::vector<float> h_output(total_size, 0.0f);
//     std::vector<float> h_grad_input(total_size, 0.0f);

//     std::vector<float> h_scale(param_size, 1.0f);  // gamma
//     std::vector<float> h_bias(param_size, 0.0f);   // beta
//     std::vector<float> h_running_mean(param_size, 0.0f);
//     std::vector<float> h_running_var(param_size, 1.0f);

//     std::vector<float> h_saved_mean(param_size);
//     std::vector<float> h_saved_var(param_size);

//     std::vector<float> h_dscale(param_size, 0.0f);
//     std::vector<float> h_dbias(param_size, 0.0f);

//     float *d_input, *d_output, *d_grad_output, *d_grad_input;
//     float *d_scale, *d_bias, *d_running_mean, *d_running_var;
//     float *d_saved_mean, *d_saved_var;
//     float *d_dscale, *d_dbias;

//     CUDA_CHECK(hipMalloc(&d_input, total_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_output, total_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_grad_output, total_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_grad_input, total_size * sizeof(float)));

//     CUDA_CHECK(hipMalloc(&d_scale, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_bias, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_running_mean, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_running_var, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_saved_mean, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_saved_var, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_dscale, param_size * sizeof(float)));
//     CUDA_CHECK(hipMalloc(&d_dbias, param_size * sizeof(float)));

//     CUDA_CHECK(hipMemcpy(d_input, h_input.data(), total_size * sizeof(float), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(d_grad_output, h_grad_output.data(), total_size * sizeof(float), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(d_scale, h_scale.data(), param_size * sizeof(float), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(d_bias, h_bias.data(), param_size * sizeof(float), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(d_running_mean, h_running_mean.data(), param_size * sizeof(float), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(d_running_var, h_running_var.data(), param_size * sizeof(float), hipMemcpyHostToDevice));

//     hipdnnTensorDescriptor_t x_desc;
//     CUDNN_CHECK(hipdnnCreateTensorDescriptor(&x_desc));
//     CUDNN_CHECK(hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

//     hipdnnTensorDescriptor_t bn_desc;
//     CUDNN_CHECK(hipdnnCreateTensorDescriptor(&bn_desc));
//     CUDNN_CHECK(hipdnnSetTensor4dDescriptor(bn_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, c, 1, 1));

//     float alpha = 1.0f, beta = 0.0f;
//     double epsilon = 1e-5;
//     double exponential_average_factor = 1.0;

//     // Forward training pass
//     CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
//         cudnn,
//         HIPDNN_BATCHNORM_SPATIAL,
//         &alpha, &beta,
//         x_desc, d_input,
//         x_desc, d_output,
//         bn_desc, d_scale, d_bias,
//         exponential_average_factor,
//         d_running_mean, d_running_var,
//         epsilon,
//         d_saved_mean, d_saved_var
//     ));

//     // Backward pass
//     CUDNN_CHECK(hipdnnBatchNormalizationBackward(
//         cudnn,
//         HIPDNN_BATCHNORM_SPATIAL,
//         &alpha, &beta,
//         &alpha, &beta,
//         x_desc, d_input,
//         x_desc, d_grad_output,
//         x_desc, d_grad_input,
//         bn_desc, d_scale,
//         d_dscale, d_dbias,
//         epsilon,
//         d_saved_mean, d_saved_var
//     ));

//     // Copy results back
//     CUDA_CHECK(hipMemcpy(h_output.data(), d_output, total_size * sizeof(float), hipMemcpyDeviceToHost));
//     CUDA_CHECK(hipMemcpy(h_grad_input.data(), d_grad_input, total_size * sizeof(float), hipMemcpyDeviceToHost));
//     CUDA_CHECK(hipMemcpy(h_dscale.data(), d_dscale, param_size * sizeof(float), hipMemcpyDeviceToHost));
//     CUDA_CHECK(hipMemcpy(h_dbias.data(), d_dbias, param_size * sizeof(float), hipMemcpyDeviceToHost));
//     CUDA_CHECK(hipMemcpy(h_saved_mean.data(), d_saved_mean, param_size * sizeof(float), hipMemcpyDeviceToHost));
//     CUDA_CHECK(hipMemcpy(h_saved_var.data(), d_saved_var, param_size * sizeof(float), hipMemcpyDeviceToHost));

//     // Expect saved mean = mean of each channel
//     ASSERT_NEAR(h_saved_mean[0], 2.0f, 1e-3);
//     ASSERT_NEAR(h_saved_mean[1], 5.0f, 1e-3);
//     ASSERT_NEAR(h_saved_var[0], 1.0f, 1e-2);  // (1+0+1)/3 = 0.6667 approx
//     ASSERT_NEAR(h_saved_var[1], 1.0f, 1e-2);

//     // Expect dBias to be 3.0 (sum of dy per channel)
//     ASSERT_NEAR(h_dbias[0], 3.0f, 1e-3);
//     ASSERT_NEAR(h_dbias[1], 3.0f, 1e-3);

//     // Expect dScale to be near 0 (since input normalized mean is 0)
//     ASSERT_NEAR(h_dscale[0], 0.0f, 1e-2);
//     ASSERT_NEAR(h_dscale[1], 0.0f, 1e-2);

//     // Cleanup
//     CUDA_CHECK(hipFree(d_input));
//     CUDA_CHECK(hipFree(d_output));
//     CUDA_CHECK(hipFree(d_grad_output));
//     CUDA_CHECK(hipFree(d_grad_input));
//     CUDA_CHECK(hipFree(d_scale));
//     CUDA_CHECK(hipFree(d_bias));
//     CUDA_CHECK(hipFree(d_running_mean));
//     CUDA_CHECK(hipFree(d_running_var));
//     CUDA_CHECK(hipFree(d_saved_mean));
//     CUDA_CHECK(hipFree(d_saved_var));
//     CUDA_CHECK(hipFree(d_dscale));
//     CUDA_CHECK(hipFree(d_dbias));
//     CUDNN_CHECK(hipdnnDestroyTensorDescriptor(x_desc));
//     CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bn_desc));
//     CUDNN_CHECK(hipdnnDestroy(cudnn));
// }

TEST(cuDNN, GetConvolution2dForwardOutputDim) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 5, w = 5;
    const int k = 1, kh = 3, kw = 3;
    const int pad = 1, stride = 1, dilation = 1;

    hipdnnTensorDescriptor_t input_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, kh, kw));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc,
        pad, pad, stride, stride, dilation, dilation,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    int out_n = 0, out_c = 0, out_h = 0, out_w = 0;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(
        conv_desc, input_desc, filter_desc,
        &out_n, &out_c, &out_h, &out_w));

    ASSERT_EQ(out_n, n);
    ASSERT_EQ(out_c, k);
    ASSERT_EQ(out_h, h);  // With padding=1, stride=1, output height = input height
    ASSERT_EQ(out_w, w);  // Same for width

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, GetConvolutionForwardAlgorithm_v7) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 5, w = 5;
    const int k = 1, kh = 3, kw = 3;
    const int pad = 1, stride = 1, dilation = 1;

    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, kh, kw));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc,
        pad, pad, stride, stride, dilation, dilation,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    int out_n, out_c, out_h, out_w;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(
        conv_desc, input_desc, filter_desc,
        &out_n, &out_c, &out_h, &out_w));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));

    int returned_algo_count = 0;
    hipdnnConvolutionFwdAlgoPerf_t perf_results[1];

    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        1,
        &returned_algo_count,
        perf_results));

    ASSERT_GE(returned_algo_count, 1);
    ASSERT_EQ(perf_results[0].status, HIPDNN_STATUS_SUCCESS);
    ASSERT_NE(perf_results[0].algo, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM);  // or just check algo is valid

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, AddTensorFloat) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 2, w = 2;
    float alpha = 2.0f, beta = 1.0f;

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

    std::vector<float> h_A = {1, 2, 3, 4};
    std::vector<float> h_B = {10, 20, 30, 40};
    std::vector<float> h_result(4);

    float *d_A, *d_B;
    CUDA_CHECK(hipMalloc(&d_A, h_A.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, h_B.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), h_A.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B.data(), h_B.size() * sizeof(float), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnAddTensor(cudnn, &alpha, desc, d_A, &beta, desc, d_B));

    CUDA_CHECK(hipMemcpy(h_result.data(), d_B, h_result.size() * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < 4; ++i) {
        ASSERT_FLOAT_EQ(h_result[i], beta * h_B[i] + alpha * h_A[i]);  // Expected: 1*B + 2*A
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, AddTensorDouble) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 2, w = 2;
    double alpha = 2.0, beta = 1.0;

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, n, c, h, w));

    std::vector<double> h_A = {1, 2, 3, 4};
    std::vector<double> h_B = {10, 20, 30, 40};
    std::vector<double> h_result(4);

    double *d_A, *d_B;
    CUDA_CHECK(hipMalloc(&d_A, h_A.size() * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, h_B.size() * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), h_A.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B.data(), h_B.size() * sizeof(double), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnAddTensor(cudnn,
                                &alpha, desc, d_A,
                                &beta, desc, d_B));

    CUDA_CHECK(hipMemcpy(h_result.data(), d_B, h_result.size() * sizeof(double), hipMemcpyDeviceToHost));

    for (int i = 0; i < 4; ++i) {
        ASSERT_DOUBLE_EQ(h_result[i], beta * h_B[i] + alpha * h_A[i]);  // Expected: 1*B + 2*A
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, TransformTensorFloat) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 2, w = 2;
    float alpha = 3.0f, beta = 0.0f;

    hipdnnTensorDescriptor_t src_desc, dest_desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&src_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dest_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(src_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dest_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

    std::vector<float> h_src = {1, 2, 3, 4};
    std::vector<float> h_dest(4, 0);

    float *d_src, *d_dest;
    CUDA_CHECK(hipMalloc(&d_src, h_src.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dest, h_dest.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_src, h_src.data(), h_src.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dest, h_dest.data(), h_dest.size() * sizeof(float), hipMemcpyHostToDevice));

    CUDNN_CHECK(cudnnTransformTensor(cudnn, &alpha, src_desc, d_src, &beta, dest_desc, d_dest));

    CUDA_CHECK(hipMemcpy(h_dest.data(), d_dest, h_dest.size() * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < 4; ++i) {
        ASSERT_FLOAT_EQ(h_dest[i], alpha * h_src[i]);  // Expected: 3 * source
    }

    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dest));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(src_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dest_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, TransformTensorDouble) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 2, w = 2;
    double alpha = 3.0, beta = 0.0;

    hipdnnTensorDescriptor_t src_desc, dest_desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&src_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dest_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(src_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, n, c, h, w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dest_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, n, c, h, w));

    std::vector<double> h_src = {1.0, 2.0, 3.0, 4.0};
    std::vector<double> h_dest(4, 0.0);

    double* d_src;
    double* d_dest;
    CUDA_CHECK(hipMalloc(&d_src, h_src.size() * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_dest, h_dest.size() * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_src, h_src.data(), h_src.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dest, h_dest.data(), h_dest.size() * sizeof(double), hipMemcpyHostToDevice));

    CUDNN_CHECK(cudnnTransformTensor(cudnn, &alpha, src_desc, d_src, &beta, dest_desc, d_dest));

    CUDA_CHECK(hipMemcpy(h_dest.data(), d_dest, h_dest.size() * sizeof(double), hipMemcpyDeviceToHost));

    for (int i = 0; i < 4; ++i) {
        ASSERT_DOUBLE_EQ(h_dest[i], alpha * h_src[i]);  // Expected: 3.0 * source
    }

    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dest));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(src_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dest_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, PoolingBackward) {
    hipdnnHandle_t cudnn;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    const int n = 1, c = 1, h = 4, w = 4;
    const int window = 2, padding = 0, stride = 2;

    hipdnnTensorDescriptor_t input_desc, output_desc, dInput_desc, dOutput_desc;
    hipdnnPoolingDescriptor_t pooling_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dInput_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dOutput_desc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pooling_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pooling_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                           window, window, padding, padding, stride, stride));

    int out_n, out_c, out_h, out_w;
    CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(pooling_desc, input_desc, &out_n, &out_c, &out_h, &out_w));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dInput_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dOutput_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

    std::vector<float> h_input(n * c * h * w, 1.0f);
    std::vector<float> h_output(out_n * out_c * out_h * out_w);
    std::vector<float> h_dOutput(out_n * out_c * out_h * out_w, 1.0f);  // Grad from next layer
    std::vector<float> h_dInput(n * c * h * w, 0.0f);

    float *d_input, *d_output, *d_dInput, *d_dOutput;
    CUDA_CHECK(hipMalloc(&d_input, h_input.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, h_output.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dInput, h_dInput.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dOutput, h_dOutput.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dOutput, h_dOutput.data(), h_dOutput.size() * sizeof(float), hipMemcpyHostToDevice));

    // Forward pooling
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc, &alpha, input_desc, d_input, &beta, output_desc, d_output));

    // Backward pooling
    CUDNN_CHECK(hipdnnPoolingBackward(cudnn, pooling_desc, &alpha, output_desc, d_output, dOutput_desc, d_dOutput, input_desc, d_input, &beta, dInput_desc, d_dInput));

    CUDA_CHECK(hipMemcpy(h_dInput.data(), d_dInput, h_dInput.size() * sizeof(float), hipMemcpyDeviceToHost));

    // Basic check: gradients should be positive and distributed in max pooling positions
    for (auto val : h_dInput) {
        ASSERT_GE(val, 0);
    }

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_dInput));
    CUDA_CHECK(hipFree(d_dOutput));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dInput_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dOutput_desc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(pooling_desc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, FindConvolutionForwardAlgorithmEx) {
    hipdnnHandle_t cudnn;
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    CUDNN_CHECK(hipdnnCreate(&cudnn));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));

    // Define tensor dimensions (NCHW)
    const int N = 1, C = 3, H = 32, W = 32;
    const int K = 16, R = 3, S = 3; // filters: output channels, input channels, filter height, filter width

    // Setup input tensor descriptor (NCHW, float)
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        inputDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        N, C, H, W));

    // Setup filter descriptor
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
        filterDesc,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        K, C, R, S));

    // Setup convolution descriptor
    const int pad_h = 1, pad_w = 1;
    const int stride_h = 1, stride_w = 1;
    const int dilation_h = 1, dilation_w = 1;
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
        convDesc,
        pad_h, pad_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));

    // Get output dimensions
    int outN, outC, outH, outW;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(
        convDesc, inputDesc, filterDesc,
        &outN, &outC, &outH, &outW));

    // Setup output tensor descriptor
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        outputDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        outN, outC, outH, outW));

    // Allocate device memory for input, filter, output
    size_t inputBytes = N * C * H * W * sizeof(float);
    size_t filterBytes = K * C * R * S * sizeof(float);
    size_t outputBytes = outN * outC * outH * outW * sizeof(float);

    float* d_input = nullptr;
    float* d_filter = nullptr;
    float* d_output = nullptr;

    CUDA_CHECK(hipMalloc(&d_input, inputBytes));
    CUDA_CHECK(hipMemset(d_input, 0, inputBytes));

    CUDA_CHECK(hipMalloc(&d_filter, filterBytes));
    CUDA_CHECK(hipMemset(d_filter, 0, filterBytes));

    CUDA_CHECK(hipMalloc(&d_output, outputBytes));
    CUDA_CHECK(hipMemset(d_output, 0, outputBytes));

    // Allocate workspace
    size_t workspaceSize = 0;
    // Just get a reasonable workspace size for one algorithm to be safe
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn,
        inputDesc,
        filterDesc,
        convDesc,
        outputDesc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        &workspaceSize));

    // Make workspace at least 1MB for safety (optional)
    workspaceSize = (workspaceSize > (1 << 20)) ? workspaceSize : (1 << 20);

    void* d_workspace = nullptr;
    CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));

    // Prepare perfResults array
    const int requestedAlgoCount = 5;
    int returnedAlgoCount = 0;
    hipdnnConvolutionFwdAlgoPerf_t perfResults[requestedAlgoCount];

    // Run the algorithm finder
    CUDNN_CHECK(hipdnnFindConvolutionForwardAlgorithmEx(
        cudnn,
        inputDesc, d_input,
        filterDesc, d_filter,
        convDesc,
        outputDesc, d_output,
        requestedAlgoCount,
        &returnedAlgoCount,
        perfResults,
        d_workspace,
        workspaceSize));

    // Checks
    ASSERT_GT(returnedAlgoCount, 0);
    for (int i = 0; i < returnedAlgoCount; i++) {
        ASSERT_GE(perfResults[i].time, 0.0f);
        ASSERT_GE(perfResults[i].memory, 0);
        ASSERT_EQ(perfResults[i].status, HIPDNN_STATUS_SUCCESS);
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_workspace));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDNN_CHECK(hipdnnDestroy(cudnn));
}

TEST(cuDNN, RNNCreateDestroyDescriptor) {
    hipdnnRNNDescriptor_t rnnDesc;
    CUDNN_CHECK(hipdnnCreateRNNDescriptor(&rnnDesc));
    CUDNN_CHECK(hipdnnDestroyRNNDescriptor(rnnDesc));
}

TEST(cuDNN, RNNCreateDestroyDataDescriptor) {
    cudnnRNNDataDescriptor_t rnnDataDesc;
    CUDNN_CHECK(cudnnCreateRNNDataDescriptor(&rnnDataDesc));
    CUDNN_CHECK(cudnnDestroyRNNDataDescriptor(rnnDataDesc));
}

TEST(cuDNN, RNNSetGetDataDescriptor) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    cudnnRNNDataDescriptor_t rnnDataDesc;
    CUDNN_CHECK(cudnnCreateRNNDataDescriptor(&rnnDataDesc));

    // Descriptor setup
    cudnnRNNDataLayout_t layout = CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_UNPACKED;
    hipdnnDataType_t dataType = HIPDNN_DATA_FLOAT;
    int maxSeqLength = 5;
    int batchSize = 3;
    int vectorSize = 8;
    std::vector<int> seqLengths = {5, 4, 3};
    float paddingFillValue = 0.0f;

    CUDNN_CHECK(cudnnSetRNNDataDescriptor(
        rnnDataDesc,
        dataType,
        layout,
        maxSeqLength,
        batchSize,
        vectorSize,
        seqLengths.data(),
        &paddingFillValue // must match `dataType` in type
    ));

    // Prepare output variables
    hipdnnDataType_t outDataType;
    cudnnRNNDataLayout_t outLayout;
    int outMaxSeqLength = 0, outBatchSize = 0, outVectorSize = 0;
    std::vector<int> outSeqLengths(batchSize);
    float outPaddingFill = -1.0f; // initial garbage value

    CUDNN_CHECK(cudnnGetRNNDataDescriptor(
        rnnDataDesc,
        &outDataType,
        &outLayout,
        &outMaxSeqLength,
        &outBatchSize,
        &outVectorSize,
        batchSize,
        outSeqLengths.data(),
        &outPaddingFill
    ));

    // Validate roundtrip
    ASSERT_EQ(outDataType, dataType);
    ASSERT_EQ(outLayout, layout);
    ASSERT_EQ(outMaxSeqLength, maxSeqLength);
    ASSERT_EQ(outBatchSize, batchSize);
    ASSERT_EQ(outVectorSize, vectorSize);
    ASSERT_FLOAT_EQ(outPaddingFill, paddingFillValue);
    for (int i = 0; i < batchSize; ++i) {
        ASSERT_EQ(outSeqLengths[i], seqLengths[i]);
    }

    CUDNN_CHECK(cudnnDestroyRNNDataDescriptor(rnnDataDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, DropoutDescriptorCreateDestroy) {
    hipdnnDropoutDescriptor_t dropoutDesc;
    CUDNN_CHECK(hipdnnCreateDropoutDescriptor(&dropoutDesc));
    CUDNN_CHECK(hipdnnDestroyDropoutDescriptor(dropoutDesc));
}

TEST(cuDNN, GetDropoutDescriptor) {
    hipdnnHandle_t handle;
    hipdnnDropoutDescriptor_t dropoutDesc;

    // Create cuDNN handle and dropout descriptor
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnCreateDropoutDescriptor(&dropoutDesc));

    // These will hold output from cudnnGetDropoutDescriptor
    float dropout;
    void* states;
    unsigned long long seed;

    // This will most likely return default/uninitialized values,
    // but the goal is to ensure it doesn't crash or return an error
    ASSERT_EQ(cudnnGetDropoutDescriptor(dropoutDesc, handle, &dropout, &states, &seed), HIPDNN_STATUS_SUCCESS);

    // Clean up
    CUDNN_CHECK(hipdnnDestroyDropoutDescriptor(dropoutDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, SetGetDropoutDescriptor) {
    hipdnnHandle_t handle;
    hipdnnDropoutDescriptor_t dropoutDesc;

    // Create cuDNN handle and dropout descriptor
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnCreateDropoutDescriptor(&dropoutDesc));

    // Allocate memory for states
    size_t stateSize;
    float dropout = 0.25f;
    unsigned long long seed = 123456789ULL;

    // Query size required for states
    CUDNN_CHECK(hipdnnDropoutGetStatesSize(handle, &stateSize));
    void* states;
    hipMalloc(&states, stateSize);

    // Set the dropout descriptor
    CUDNN_CHECK(hipdnnSetDropoutDescriptor(dropoutDesc, handle, dropout, states, stateSize, seed));

    // Get the descriptor values
    float returned_dropout;
    void* returned_states;
    unsigned long long returned_seed;

    CUDNN_CHECK(cudnnGetDropoutDescriptor(dropoutDesc, handle, &returned_dropout, &returned_states, &returned_seed));

    // Assert values are correct
    ASSERT_FLOAT_EQ(returned_dropout, dropout);
    ASSERT_EQ(returned_seed, seed);
    ASSERT_EQ(returned_states, states);

    // Cleanup
    hipFree(states);
    CUDNN_CHECK(hipdnnDestroyDropoutDescriptor(dropoutDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, RNNForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    // RNN config
    const int inputSize = 8;
    const int hiddenSize = 16;
    const int numLayers = 1;
    const int batchSize = 2;
    const int seqLength = 4;
    const int projSize = hiddenSize;

    hipdnnRNNDescriptor_t rnnDesc;
    CUDNN_CHECK(hipdnnCreateRNNDescriptor(&rnnDesc));

    hipdnnDropoutDescriptor_t dropoutDesc;
    CUDNN_CHECK(hipdnnCreateDropoutDescriptor(&dropoutDesc));

    // Dummy dropout config (no dropout)
    void* states;
    size_t stateSize;
    CUDNN_CHECK(hipdnnDropoutGetStatesSize(handle, &stateSize));
    CUDA_CHECK(hipMalloc(&states, stateSize));
    CUDNN_CHECK(hipdnnSetDropoutDescriptor(
        dropoutDesc, handle, 0.0f, states, stateSize, 0));

    CUDNN_CHECK(cudnnSetRNNDescriptor_v8(
        rnnDesc,
        HIPDNN_RNN_ALGO_STANDARD,
        HIPDNN_LSTM,
        HIPDNN_RNN_WITH_BIAS,
        HIPDNN_UNIDIRECTIONAL,
        HIPDNN_LINEAR_INPUT,
        HIPDNN_DATA_FLOAT,
        HIPDNN_DATA_FLOAT,
        HIPDNN_DEFAULT_MATH,
        inputSize,
        hiddenSize,
        projSize,
        numLayers,
        dropoutDesc,
        0));

    // Input and output RNNDataDescriptors
    cudnnRNNDataDescriptor_t xDesc, yDesc;
    CUDNN_CHECK(cudnnCreateRNNDataDescriptor(&xDesc));
    CUDNN_CHECK(cudnnCreateRNNDataDescriptor(&yDesc));

    int seqLengthArray[batchSize];
    for (int i = 0; i < batchSize; ++i) seqLengthArray[i] = seqLength;

    CUDNN_CHECK(cudnnSetRNNDataDescriptor(
        xDesc, HIPDNN_DATA_FLOAT, CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED,
        seqLength, batchSize, inputSize,
        seqLengthArray, nullptr));

    CUDNN_CHECK(cudnnSetRNNDataDescriptor(
        yDesc, HIPDNN_DATA_FLOAT, CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED,
        seqLength, batchSize, hiddenSize,
        seqLengthArray, nullptr));

    // Allocate dummy input/output memory
    size_t inputBytes = seqLength * batchSize * inputSize * sizeof(float);
    size_t outputBytes = seqLength * batchSize * hiddenSize * sizeof(float);
    float* x; float* y;
    CUDA_CHECK(hipMalloc(&x, inputBytes));
    CUDA_CHECK(hipMalloc(&y, outputBytes));

    // Initial/Final hidden/cell states
    hipdnnTensorDescriptor_t hDesc, cDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&hDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&cDesc));

    int dims[3] = {numLayers, batchSize, hiddenSize};
    int strides[3] = {batchSize * hiddenSize, hiddenSize, 1};
    CUDNN_CHECK(hipdnnSetTensorNdDescriptor(hDesc, HIPDNN_DATA_FLOAT, 3, dims, strides));
    CUDNN_CHECK(hipdnnSetTensorNdDescriptor(cDesc, HIPDNN_DATA_FLOAT, 3, dims, strides));

    float *hx, *cx, *hy, *cy;
    CUDA_CHECK(hipMalloc(&hx, numLayers * batchSize * hiddenSize * sizeof(float)));
    CUDA_CHECK(hipMalloc(&cx, numLayers * batchSize * hiddenSize * sizeof(float)));
    CUDA_CHECK(hipMalloc(&hy, numLayers * batchSize * hiddenSize * sizeof(float)));
    CUDA_CHECK(hipMalloc(&cy, numLayers * batchSize * hiddenSize * sizeof(float)));

    // Allocate weights
    size_t weightSpaceSize;
    CUDNN_CHECK(cudnnGetRNNWeightSpaceSize(handle, rnnDesc, &weightSpaceSize));
    void* weightSpace;
    CUDA_CHECK(hipMalloc(&weightSpace, weightSpaceSize));

    // Workspace and reserve space
    size_t workspaceSize, reserveSize;
    CUDNN_CHECK(cudnnGetRNNTempSpaceSizes(
        handle, rnnDesc, CUDNN_FWD_MODE_TRAINING,
        xDesc, &workspaceSize, &reserveSize));
    void* workspace; void* reserveSpace;
    CUDA_CHECK(hipMalloc(&workspace, workspaceSize));
    CUDA_CHECK(hipMalloc(&reserveSpace, reserveSize));

    // Allocate devSeqLengths
    int32_t* devSeqLengths;
    CUDA_CHECK(hipMalloc(&devSeqLengths, batchSize * sizeof(int32_t)));
    CUDA_CHECK(hipMemcpy(devSeqLengths, seqLengthArray,
                          batchSize * sizeof(int32_t), hipMemcpyHostToDevice));

    // Run forward
    CUDNN_CHECK(cudnnRNNForward(
        handle, rnnDesc, CUDNN_FWD_MODE_TRAINING, devSeqLengths,
        xDesc, x, yDesc, y,
        hDesc, hx, hy, cDesc, cx, cy,
        weightSpaceSize, weightSpace,
        workspaceSize, workspace,
        reserveSize, reserveSpace));

    // Cleanup
    CUDA_CHECK(hipFree(x));
    CUDA_CHECK(hipFree(y));
    CUDA_CHECK(hipFree(hx));
    CUDA_CHECK(hipFree(hy));
    CUDA_CHECK(hipFree(cx));
    CUDA_CHECK(hipFree(cy));
    CUDA_CHECK(hipFree(weightSpace));
    CUDA_CHECK(hipFree(workspace));
    CUDA_CHECK(hipFree(reserveSpace));
    CUDA_CHECK(hipFree(states));
    CUDA_CHECK(hipFree(devSeqLengths));

    CUDNN_CHECK(hipdnnDestroyRNNDescriptor(rnnDesc));
    CUDNN_CHECK(hipdnnDestroyDropoutDescriptor(dropoutDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(hDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(cDesc));
    CUDNN_CHECK(cudnnDestroyRNNDataDescriptor(xDesc));
    CUDNN_CHECK(cudnnDestroyRNNDataDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

// Not implemented yet
// TEST(cuDNN, FusedOpsPlanCreateDestroy) {
//     cudnnFusedOpsPlan_t plan;

//     // Choose a valid fused op. For example: convolution + bias + activation
//     cudnnFusedOps_t ops = CUDNN_FUSED_SCALE_BIAS_ACTIVATION_CONV_BNSTATS;

//     // Create the fused ops plan
//     CUDNN_CHECK(cudnnCreateFusedOpsPlan(&plan, ops));

//     // Now destroy the plan if it was created
//     CUDNN_CHECK(cudnnDestroyFusedOpsPlan(plan));
// }

TEST(cuDNN, createDestroyLRNDescriptor) {
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
}

TEST(cuDNN, SetGetLRNDescriptor) {
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));

    // Set LRN descriptor
    unsigned n = 5;  // Local size
    double alpha = 1.0f, beta = 0.75f, k = 1e-4f;
    CUDNN_CHECK(hipdnnSetLRNDescriptor(lrnDesc, n, alpha, beta, k));

    // Get LRN descriptor
    unsigned out_n;
    double out_alpha, out_beta, out_k;
    CUDNN_CHECK(hipdnnGetLRNDescriptor(lrnDesc, &out_n, &out_alpha, &out_beta, &out_k));

    ASSERT_EQ(out_n, n);
    ASSERT_FLOAT_EQ(out_alpha, alpha);
    ASSERT_FLOAT_EQ(out_beta, beta);
    ASSERT_FLOAT_EQ(out_k, k);

    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
}

TEST(cuDNN, LRNCrossChannelForwardFloat) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    const int N = 1, C = 3, H = 5, W = 5;
    const size_t tensorSize = N * C * H * W;

    // Allocate host and device memory
    std::vector<float> h_input(tensorSize, 1.0f);  // simple input
    std::vector<float> h_output(tensorSize, 0.0f);

    float *d_input = nullptr;
    float *d_output = nullptr;
    CUDA_CHECK(hipMalloc(&d_input, tensorSize * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, tensorSize * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), tensorSize * sizeof(float), hipMemcpyHostToDevice));

    // Create and set tensor descriptors
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

    // Create and set LRN descriptor
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));
    unsigned lrnN = 5;
    double lrnAlpha = 1e-4;
    double lrnBeta = 0.75;
    double lrnK = 2.0;
    CUDNN_CHECK(hipdnnSetLRNDescriptor(lrnDesc, lrnN, lrnAlpha, lrnBeta, lrnK));

    // Perform forward pass
    const float alpha = 1.0f;
    const float beta = 0.0f;
    CUDNN_CHECK(hipdnnLRNCrossChannelForward(handle, lrnDesc, HIPDNN_LRN_CROSS_CHANNEL,
                                            &alpha, desc, d_input,
                                            &beta, desc, d_output));

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, tensorSize * sizeof(float), hipMemcpyDeviceToHost));

    // Optional: Validate results are different from input
    for (size_t i = 0; i < tensorSize; ++i) {
        ASSERT_NE(h_output[i], 0.0f);  // since input was all 1s, LRN will produce something non-zero
    }

    // Cleanup
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
}

TEST(cuDNN, LRNCrossChannelForwardDouble) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    const int N = 1, C = 3, H = 5, W = 5;
    const size_t tensorSize = N * C * H * W;

    // Allocate host and device memory
    std::vector<double> h_input(tensorSize, 1.0);  // simple input
    std::vector<double> h_output(tensorSize, 0.0);

    double* d_input = nullptr;
    double* d_output = nullptr;
    CUDA_CHECK(hipMalloc(&d_input, tensorSize * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_output, tensorSize * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), tensorSize * sizeof(double), hipMemcpyHostToDevice));

    // Create and set tensor descriptors
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W));

    // Create and set LRN descriptor
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));
    unsigned lrnN = 5;
    double lrnAlpha = 1e-4;
    double lrnBeta = 0.75;
    double lrnK = 2.0;
    CUDNN_CHECK(hipdnnSetLRNDescriptor(lrnDesc, lrnN, lrnAlpha, lrnBeta, lrnK));

    // Perform forward pass
    const double alpha = 1.0;
    const double beta = 0.0;
    CUDNN_CHECK(hipdnnLRNCrossChannelForward(handle, lrnDesc, HIPDNN_LRN_CROSS_CHANNEL,
                                            &alpha, desc, d_input,
                                            &beta, desc, d_output));

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, tensorSize * sizeof(double), hipMemcpyDeviceToHost));

    // Optional: Validate results are different from input
    for (size_t i = 0; i < tensorSize; ++i) {
        ASSERT_NE(h_output[i], 0.0);  // since input was all 1s, LRN will produce something non-zero
    }

    // Cleanup
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
}
