/*
 * Written By: Theodoros Aslanidis <theodoros.aslanidis@ucdconnect.ie>
 *             School of Computer Science, University College Dublin
 */

#include <gtest/gtest.h>
#include <hip/hiprtc.h>

#define NVRTC_CHECK(err) ASSERT_EQ((err), HIPRTC_SUCCESS)

TEST(nvrtc, versionCheck) {
    int major = 0, minor = 0;
    hiprtcVersion(&major, &minor);
    ASSERT_EQ(major, 0);  // I have not implemented this, so it should be 0
    ASSERT_EQ(minor, 0);  // I have not implemented this, so it should be 0
}
