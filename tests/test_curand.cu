#include "hip/hip_runtime.h"
/*
 * Written By: Theodoros Aslanidis <theodoros.aslanidis@ucdconnect.ie>
 *             School of Computer Science, University College Dublin
 */

#include <hiprand.h>
#include <gtest/gtest.h>

#include <iostream>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA error: " << hipGetErrorString(err)
#define CURAND_CHECK(err) ASSERT_EQ((err), HIPRAND_STATUS_SUCCESS)

TEST(cuRAND, CreateDestroyGenerator) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandDestroyGenerator(generator));
}

TEST(cuRAND, CreateDestroyGeneratorHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandDestroyGenerator(generator));
}

TEST(cuRAND, SetSeed) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));
    CURAND_CHECK(hiprandDestroyGenerator(generator));
}

TEST(cuRAND, GenerateDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));

    const size_t n = 10;
    unsigned int* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(unsigned int)));

    CURAND_CHECK(hiprandGenerate(generator, output, n));

    unsigned int host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(unsigned int), hipMemcpyDeviceToHost));

    bool all_zero = true;
    for (size_t i = 0; i < n; ++i) {
        if (host_output[i] != 0) {
            all_zero = false;
            break;
        }
    }
    ASSERT_FALSE(all_zero);  // Generated numbers should not all be zero

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));

    const size_t n = 10;
    unsigned int* output = (unsigned int*)malloc(n * sizeof(unsigned int));
    ASSERT_NE(output, nullptr);

    CURAND_CHECK(hiprandGenerate(generator, output, n));

    bool all_zero = true;
    for (size_t i = 0; i < n; ++i) {
        if (output[i] != 0) {
            all_zero = false;
            break;
        }
    }
    ASSERT_FALSE(all_zero);  // Generated numbers should not all be zero

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GenerateLongLongDevice) {
    hiprandGenerator_t generator;
    const size_t num = 10;

    // Create a quasi-random number generator
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SOBOL64));

    // Set dimensions (required for quasi generators)
    CURAND_CHECK(hiprandSetQuasiRandomGeneratorDimensions(generator, 1));

    // Allocate device memory
    unsigned long long* d_output = nullptr;
    CUDA_CHECK(hipMalloc(&d_output, num * sizeof(unsigned long long)));

    // Generate quasi-random numbers
    CURAND_CHECK(hiprandGenerateLongLong(generator, d_output, num));

    // Copy results back to host for checking
    unsigned long long h_output[num];
    CUDA_CHECK(
        hipMemcpy(h_output, d_output, num * sizeof(unsigned long long), hipMemcpyDeviceToHost));

    // Clean up
    CUDA_CHECK(hipFree(d_output));
    CURAND_CHECK(hiprandDestroyGenerator(generator));
}

TEST(cuRAND, GenerateLongLongHost) {
    hiprandGenerator_t generator;
    const size_t num = 10;

    // Create a QUASI-random number generator (host generator)
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_QUASI_SOBOL64));

    // Set dimensions (required for quasi generators)
    CURAND_CHECK(hiprandSetQuasiRandomGeneratorDimensions(generator, 1));

    // Allocate host memory for output
    unsigned long long* h_output = new unsigned long long[num];

    // Generate random numbers on host
    CURAND_CHECK(hiprandGenerateLongLong(generator, h_output, num));

    delete[] h_output;
    CURAND_CHECK(hiprandDestroyGenerator(generator));
}

TEST(cuRAND, GenerateUniformDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));

    const size_t n = 10;
    float* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(float)));

    CURAND_CHECK(hiprandGenerateUniform(generator, output, n));

    float host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost));

    for (size_t i = 0; i < n; ++i) {
        ASSERT_GE(host_output[i], 0.0f);
        ASSERT_LT(host_output[i], 1.0f);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateUniformHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));

    const size_t n = 10;
    float* output = (float*)malloc(n * sizeof(float));
    ASSERT_NE(output, nullptr);

    CURAND_CHECK(hiprandGenerateUniform(generator, output, n));

    for (size_t i = 0; i < n; ++i) {
        ASSERT_GE(output[i], 0.0f);
        ASSERT_LT(output[i], 1.0f);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GenerateNormalDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 9012ULL));

    const size_t n = 1000;  // Larger sample for stats
    float* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(float)));

    const float mean = 5.0f;
    const float stddev = 2.0f;
    CURAND_CHECK(hiprandGenerateNormal(generator, output, n, mean, stddev));

    float host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost));

    // Basic sanity: check mean and stddev roughly
    float sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        sum += host_output[i];
    }
    float sample_mean = sum / n;
    ASSERT_NEAR(sample_mean, mean, 0.2f);

    float variance_sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        float diff = host_output[i] - mean;
        variance_sum += diff * diff;
    }
    float sample_stddev = sqrt(variance_sum / n);
    ASSERT_NEAR(sample_stddev, stddev, 0.3f);

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateNormalHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 9012ULL));

    const size_t n = 1000;
    float* output = (float*)malloc(n * sizeof(float));
    ASSERT_NE(output, nullptr);

    const float mean = 5.0f;
    const float stddev = 2.0f;
    CURAND_CHECK(hiprandGenerateNormal(generator, output, n, mean, stddev));

    // Basic sanity: check mean and stddev roughly
    float sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        sum += output[i];
    }
    float sample_mean = sum / n;
    ASSERT_NEAR(sample_mean, mean, 0.2f);

    float variance_sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        float diff = output[i] - mean;
        variance_sum += diff * diff;
    }
    float sample_stddev = sqrt(variance_sum / n);
    ASSERT_NEAR(sample_stddev, stddev, 0.3f);

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GenerateLogNormalDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 8642ULL));

    const size_t n = 1000;
    float* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(float)));

    const float mean = 0.0f;
    const float stddev = 0.5f;
    CURAND_CHECK(hiprandGenerateLogNormal(generator, output, n, mean, stddev));

    float host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost));

    // All values should be positive
    for (size_t i = 0; i < n; ++i) {
        ASSERT_GT(host_output[i], 0.0f);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateLogNormalHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 8642ULL));

    const size_t n = 1000;
    float* output = (float*)malloc(n * sizeof(float));
    ASSERT_NE(output, nullptr);

    const float mean = 0.0f;
    const float stddev = 0.5f;
    CURAND_CHECK(hiprandGenerateLogNormal(generator, output, n, mean, stddev));

    // All values should be positive
    for (size_t i = 0; i < n; ++i) {
        ASSERT_GT(output[i], 0.0f);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GeneratePoissonDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 3456ULL));

    const size_t n = 1000;
    unsigned int* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(unsigned int)));

    double lambda = 4.5;
    CURAND_CHECK(hiprandGeneratePoisson(generator, output, n, lambda));

    unsigned int host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // Basic sanity checks: values >= 0 and mean close to lambda
    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        ASSERT_GE(host_output[i], 0u);
        sum += host_output[i];
    }
    double sample_mean = sum / n;
    ASSERT_NEAR(sample_mean, lambda, 0.2);

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GeneratePoissonHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 3456ULL));

    const size_t n = 1000;
    unsigned int* output = (unsigned int*)malloc(n * sizeof(unsigned int));
    ASSERT_NE(output, nullptr);

    double lambda = 4.5;
    CURAND_CHECK(hiprandGeneratePoisson(generator, output, n, lambda));

    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        ASSERT_GE(output[i], 0u);
        sum += output[i];
    }
    double sample_mean = sum / n;
    ASSERT_NEAR(sample_mean, lambda, 0.2);

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GenerateUniformDoubleDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 7890ULL));

    const size_t n = 10;
    double* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(double)));

    CURAND_CHECK(hiprandGenerateUniformDouble(generator, output, n));

    double host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(double), hipMemcpyDeviceToHost));

    for (size_t i = 0; i < n; ++i) {
        ASSERT_GE(host_output[i], 0.0);
        ASSERT_LT(host_output[i], 1.0);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateUniformDoubleHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 7890ULL));

    const size_t n = 10;
    double* output = (double*)malloc(n * sizeof(double));
    ASSERT_NE(output, nullptr);

    CURAND_CHECK(hiprandGenerateUniformDouble(generator, output, n));

    for (size_t i = 0; i < n; ++i) {
        ASSERT_GE(output[i], 0.0);
        ASSERT_LT(output[i], 1.0);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GenerateNormalDoubleDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 2468ULL));

    const size_t n = 1000;
    double* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(double)));

    const double mean = 10.0;
    const double stddev = 3.0;
    CURAND_CHECK(hiprandGenerateNormalDouble(generator, output, n, mean, stddev));

    double host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(double), hipMemcpyDeviceToHost));

    // Basic sanity: check mean and stddev roughly
    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        sum += host_output[i];
    }
    double sample_mean = sum / n;
    ASSERT_NEAR(sample_mean, mean, 0.2);

    double variance_sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        double diff = host_output[i] - mean;
        variance_sum += diff * diff;
    }
    double sample_stddev = sqrt(variance_sum / n);
    ASSERT_NEAR(sample_stddev, stddev, 0.3);

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateNormalDoubleHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 2468ULL));

    const size_t n = 1000;
    double* output = (double*)malloc(n * sizeof(double));
    ASSERT_NE(output, nullptr);

    const double mean = 10.0;
    const double stddev = 3.0;
    CURAND_CHECK(hiprandGenerateNormalDouble(generator, output, n, mean, stddev));

    // Basic sanity: check mean and stddev roughly
    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        sum += output[i];
    }
    double sample_mean = sum / n;
    ASSERT_NEAR(sample_mean, mean, 0.2);

    double variance_sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        double diff = output[i] - mean;
        variance_sum += diff * diff;
    }
    double sample_stddev = sqrt(variance_sum / n);
    ASSERT_NEAR(sample_stddev, stddev, 0.3);

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}

TEST(cuRAND, GenerateLogNormalDoubleDevice) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1357ULL));

    const size_t n = 1000;
    double* output;
    CUDA_CHECK(hipMalloc(&output, n * sizeof(double)));

    const double mean = 0.0;    // mean of underlying normal
    const double stddev = 0.5;  // stddev of underlying normal
    CURAND_CHECK(hiprandGenerateLogNormalDouble(generator, output, n, mean, stddev));

    double host_output[n];
    CUDA_CHECK(hipMemcpy(host_output, output, n * sizeof(double), hipMemcpyDeviceToHost));

    // All outputs should be positive
    for (size_t i = 0; i < n; ++i) {
        ASSERT_GT(host_output[i], 0.0);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    CUDA_CHECK(hipFree(output));
}

TEST(cuRAND, GenerateLogNormalDoubleHost) {
    hiprandGenerator_t generator;
    CURAND_CHECK(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, 1357ULL));

    const size_t n = 1000;
    double* output = (double*)malloc(n * sizeof(double));
    ASSERT_NE(output, nullptr);

    const double mean = 0.0;
    const double stddev = 0.5;
    CURAND_CHECK(hiprandGenerateLogNormalDouble(generator, output, n, mean, stddev));

    // All outputs should be positive
    for (size_t i = 0; i < n; ++i) {
        ASSERT_GT(output[i], 0.0);
    }

    CURAND_CHECK(hiprandDestroyGenerator(generator));
    free(output);
}