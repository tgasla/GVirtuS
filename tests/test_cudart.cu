#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> /* cuuint64_t */

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)

__device__ int intDeviceVariable = 0;

TEST(cudaRT, getDeviceCount) {
    int count = 0;
    CUDA_CHECK(hipGetDeviceCount(&count));
    ASSERT_GT(count, 0);
}

TEST(cudaRT, ThreadExchangeStreamCaptureMode) {
    hipStreamCaptureMode original_mode = hipStreamCaptureModeThreadLocal;
    hipStreamCaptureMode previous_mode;

    // Exchange thread-local with current mode, original_mode gets overwritten with previous
    previous_mode = original_mode;
    CUDA_CHECK(hipThreadExchangeStreamCaptureMode(&original_mode));

    // Ensure that exchange actually happened: value at `original_mode` now holds the previous
    ASSERT_NE(previous_mode, original_mode);

    // Now push the original mode back to restore thread state
    CUDA_CHECK(hipThreadExchangeStreamCaptureMode(&original_mode));

    // Ensure that the original mode is restored
    ASSERT_EQ(original_mode, previous_mode);
}

TEST(cudaRT, MemPoolGetAttribute) {
    hipMemPool_t memPool;
    CUDA_CHECK(hipDeviceGetDefaultMemPool(&memPool, 0));

    cuuint64_t threshold = 0;
    CUDA_CHECK(hipMemPoolGetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &threshold));
}

TEST(cudaRT, MallocFree) {
    void* devPtr = nullptr;
    CUDA_CHECK(hipMalloc(&devPtr, 1024));
    CUDA_CHECK(hipFree(devPtr));
}

TEST(cudaRT, MemcpySync) {
    int h_src = 42;
    int h_dst = 0;
    int* d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_ptr, &h_src, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&h_dst, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
    ASSERT_EQ(h_dst, 42);

    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, MemcpyAsync) {
    int h_src = 24;
    int h_dst = 0;
    int* d_ptr;
    hipStream_t stream;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDA_CHECK(hipMemcpyAsync(d_ptr, &h_src, sizeof(int), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_dst, d_ptr, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(h_dst, 24);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, Memset) {
    int* d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));
    CUDA_CHECK(hipMemset(d_ptr, 0, sizeof(int)));

    int h_val = 1;
    CUDA_CHECK(hipMemcpy(&h_val, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
    ASSERT_EQ(h_val, 0);

    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, StreamCreateDestroySynchronize) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(cudaRT, StreamCaptureBeginEnd) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    hipStreamCaptureMode mode = hipStreamCaptureModeThreadLocal;
    CUDA_CHECK(hipStreamBeginCapture(stream, mode));
    hipGraph_t graph;
    CUDA_CHECK(hipStreamEndCapture(stream, &graph));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(cudaRT, GraphCreateDestroy) {
    hipGraph_t graph;
    CUDA_CHECK(hipGraphCreate(&graph, 0));
    CUDA_CHECK(hipGraphDestroy(graph));
}

TEST(cudaRT, GraphInstantiateDestroy) {
    hipGraph_t graph;
    CUDA_CHECK(hipGraphCreate(&graph, 0));
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, 0));
    CUDA_CHECK(hipGraphDestroy(graph));
}

TEST(cudaRT, GetDevice) {
    int device;
    CUDA_CHECK(hipGetDevice(&device));
}

TEST(cudaRT, SetDevice) {
    int device = 0;
    CUDA_CHECK(hipSetDevice(device));
}

TEST(cudaRT, DeviceSynchronize) {
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void simpleKernel(int* output) {
    *output = 123;
}

TEST(cudaRT, LaunchKernel) {
    int* d_output;
    CUDA_CHECK(hipMalloc(&d_output, sizeof(int)));
    CUDA_CHECK(hipMemset(d_output, 0, sizeof(int)));

    void* args[] = { &d_output };

    dim3 grid(1), block(1);
    CUDA_CHECK(hipLaunchKernel((const void*)simpleKernel,
                                grid, block,
                                args,
                                0, nullptr));

    int h_output = 0;
    CUDA_CHECK(hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost));
    ASSERT_EQ(h_output, 123);

    CUDA_CHECK(hipFree(d_output));
}

TEST(cudaRT, PushCallConfiguration) {
    dim3 grid(1), block(1);
    size_t shared = 0;
    hipStream_t stream = 0;
    CUDA_CHECK(__cudaPushCallConfiguration(grid, block, shared, stream));
}

TEST(CudaRT, KernelLaunchWithTripletSyntax) {
    int* d_out = nullptr;
    int h_out = 0;

    // Allocate memory on device
    CUDA_CHECK(hipMalloc(&d_out, sizeof(int)));

    // Launch kernel with <<<>>> syntax
    simpleKernel<<<1, 1>>>(d_out);

    // Wait for kernel to complete
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost));

    // Verify kernel result
    ASSERT_EQ(h_out, 123);

    CUDA_CHECK(hipFree(d_out));
}

TEST(cudaRT, EventCreateRecordSynchronizeElapsedTimeDestroy) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float elapsed_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
    ASSERT_GT(elapsed_ms, 0.0f);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}
