#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA error: " << hipGetErrorString(err)
#define CUBLAS_CHECK(err) ASSERT_EQ((err), HIPBLAS_STATUS_SUCCESS)

TEST(cuBLAS, CreateDestroy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    ASSERT_NE(handle, nullptr);
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, GetVersion) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    int version;
    CUBLAS_CHECK(cublasGetVersion(handle, &version));
    ASSERT_GT(version, 0); // Ensure version is a positive integer
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, SetStreamDestroy) {
    hipblasHandle_t handle;
    hipStream_t stream;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUBLAS_CHECK(hipblasSetStream(handle, stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Sgemm) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Matrix size 2x2 for simplicity
    const int N = 2;
    float h_A[] = {1, 2, 3, 4};  // col-major 2x2
    float h_B[] = {5, 6, 7, 8};
    float h_C[4] = {0};

    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, N * N * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_C, 0, N * N * sizeof(float)));

    float alpha = 1.0f, beta = 0.0f;

    // SGEMM: C = alpha * A * B + beta * C
    CUBLAS_CHECK(hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha,
                             d_A, N,
                             d_B, N,
                             &beta,
                             d_C, N));

    CUDA_CHECK(hipMemcpy(h_C, d_C, sizeof(h_C), hipMemcpyDeviceToHost));

    // Check a few expected values
    // Let's calculate correct expected values for col-major:
    // C = A * B, with A and B col-major:
    // A = |1 3|
    //     |2 4|
    // B = |5 7|
    //     |6 8|
    // C[0,0] = 1*5 + 3*6 = 5 + 18 = 23
    // C[1,0] = 2*5 + 4*6 = 10 + 24 = 34
    // C[0,1] = 1*7 + 3*8 = 7 + 24 = 31
    // C[1,1] = 2*7 + 4*8 = 14 + 32 = 46

    ASSERT_FLOAT_EQ(h_C[0], 23.0f); // 1*5+3*6=5+18=23 (wait: col-major, careful!)
    ASSERT_FLOAT_EQ(h_C[1], 34.0f);
    ASSERT_FLOAT_EQ(h_C[2], 31.0f);
    ASSERT_FLOAT_EQ(h_C[3], 46.0f);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Sgemv) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    const int M = 2, N = 2;
    float h_A[] = {1, 3, 2, 4};  // 2x2 col-major
    float h_x[] = {1, 2};
    float h_y[] = {0, 0};

    float *d_A, *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_A, M * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_x, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, M * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_y, 0, M * sizeof(float)));

    float alpha = 1.0f, beta = 0.0f;

    CUBLAS_CHECK(hipblasSgemv(handle,
                             HIPBLAS_OP_N,
                             M, N,
                             &alpha,
                             d_A, M,
                             d_x, 1,
                             &beta,
                             d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, M * sizeof(float), hipMemcpyDeviceToHost));

//     y[0] = 1*1 + 2*2 = 1 + 4 = 5
//     y[1] = 3*1 + 4*2 = 3 + 8 = 11
    ASSERT_FLOAT_EQ(h_y[0], 5.0f);
    ASSERT_FLOAT_EQ(h_y[1], 11.0f);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Saxpy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {1, 2, 3};
    float h_y[] = {4, 5, 6};

    float *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    float alpha = 2.0f;

    CUBLAS_CHECK(hipblasSaxpy(handle, n, &alpha, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    // y = y + alpha*x = [4+2*1,5+2*2,6+2*3] = [6,9,12]
    ASSERT_FLOAT_EQ(h_y[0], 6.0f);
    ASSERT_FLOAT_EQ(h_y[1], 9.0f);
    ASSERT_FLOAT_EQ(h_y[2], 12.0f);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Scopy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {1, 2, 3};
    float h_y[] = {0, 0, 0};

    float *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasScopy(handle, n, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; ++i) {
        ASSERT_FLOAT_EQ(h_y[i], h_x[i]);
    }

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Snrm2) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {3, 4, 0};

    float *d_x;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));

    float result = 0;
    CUBLAS_CHECK(hipblasSnrm2(handle, n, d_x, 1, &result));

    ASSERT_NEAR(result, 5.0f, 1e-5);

    CUDA_CHECK(hipFree(d_x));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Sdot) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {1, 2, 3};
    float h_y[] = {4, 5, 6};

    float *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    float result = 0;
    CUBLAS_CHECK(hipblasSdot(handle, n, d_x, 1, d_y, 1, &result));

    // dot product = 1*4 + 2*5 + 3*6 = 32
    ASSERT_FLOAT_EQ(result, 32.0f);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dgemm) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int m = 2, n = 3, k = 2;
    double alpha = 1.0, beta = 0.0;

    // A is m x k
    double h_A[] = {1.0, 3.0,
                    2.0, 4.0};

    // B is k x n
    double h_B[] = {5.0, 6.0, 7.0,
                    8.0, 9.0, 10.0};

    // C is m x n
    double h_C[6] = {0};

    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, m * k * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, k * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, m * n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, sizeof(h_C), hipMemcpyHostToDevice));

    // Perform C = alpha * A * B + beta * C
    // Note: cuBLAS is column-major by default, so we can either transpose matrices or switch order:
    // Using row-major layout, specify operation flags as HIPBLAS_OP_T to transpose inputs

    // Here, to keep it simple, we use the matrices as column-major:
    CUBLAS_CHECK(hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             m, n, k,
                             &alpha,
                             d_A, m,
                             d_B, k,
                             &beta,
                             d_C, m));

    CUDA_CHECK(hipMemcpy(h_C, d_C, sizeof(h_C), hipMemcpyDeviceToHost));

    double expected[] = {17, 39, 23, 53, 29, 67};
    for (int i = 0; i < m * n; ++i) {
        ASSERT_NEAR(h_C[i], expected[i], 1e-9);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dgemv) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int m = 2, n = 3;
    double alpha = 1.0, beta = 0.0;

    // A is m x n
    double h_A[] = {1.0, 2.0, 3.0,
                    4.0, 5.0, 6.0};

    double h_x[] = {1.0, 1.0, 1.0};
    double h_y[] = {0.0, 0.0};

    double *d_A, *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_A, m * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, m * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    // y = alpha * A * x + beta * y
    CUBLAS_CHECK(hipblasDgemv(handle,
                             HIPBLAS_OP_N,
                             m, n,
                             &alpha,
                             d_A, m,
                             d_x, 1,
                             &beta,
                             d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    // Expected result:
    // y[0] = 1*1 + 3*1 + 5*1 = 9
    // y[1] = 2*1 + 4*1 + 6*1 = 12

    ASSERT_NEAR(h_y[0], 9.0, 1e-9);
    ASSERT_NEAR(h_y[1], 12.0, 1e-9);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(CuBLAS, Daxpy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double alpha = 2.0;
    double h_x[] = {1.0, 2.0, 3.0};
    double h_y[] = {4.0, 5.0, 6.0};

    double *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    // y = alpha * x + y
    CUBLAS_CHECK(hipblasDaxpy(handle, n, &alpha, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    // Expected y = [4+2*1, 5+2*2, 6+2*3] = [6, 9, 12]
    ASSERT_NEAR(h_y[0], 6.0, 1e-9);
    ASSERT_NEAR(h_y[1], 9.0, 1e-9);
    ASSERT_NEAR(h_y[2], 12.0, 1e-9);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dcopy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double h_x[] = {1.0, 2.0, 3.0};
    double h_y[] = {0.0, 0.0, 0.0};

    double *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasDcopy(handle, n, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; ++i) {
        ASSERT_DOUBLE_EQ(h_y[i], h_x[i]);
    }

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dnrm2) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double h_x[] = {3.0, 4.0, 0.0};

    double *d_x;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));

    double result = 0;
    CUBLAS_CHECK(hipblasDnrm2(handle, n, d_x, 1, &result));

    ASSERT_NEAR(result, 5.0, 1e-9);

    CUDA_CHECK(hipFree(d_x));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Ddot) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double h_x[] = {1.0, 2.0, 3.0};
    double h_y[] = {4.0, 5.0, 6.0};

    double *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    double result = 0;
    CUBLAS_CHECK(hipblasDdot(handle, n, d_x, 1, d_y, 1, &result));

    // dot product = 1*4 + 2*5 + 3*6 = 32
    ASSERT_DOUBLE_EQ(result, 32.0);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, SetWorkspace) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    size_t workspaceSize = 1 << 20; // 1 MB workspace
    void* d_workspace;
    CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));

    // Attach GPU workspace to the cuBLAS handle
    CUBLAS_CHECK(hipblasSetWorkspace(handle, d_workspace, workspaceSize));

    CUDA_CHECK(hipFree(d_workspace));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLASLt, MatmulDescCreateDestroy) {
    hipblasLtMatmulDesc_t matmulDesc;
    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    // Check if the descriptor was created successfully
    ASSERT_NE(matmulDesc, nullptr);

    // Destroy the descriptor
    CUBLAS_CHECK(hipblasLtMatmulDescDestroy(matmulDesc));
}

TEST(cuBLASLt, MatmulDescSetAttribute) {
    hipblasLtMatmulDesc_t matmulDesc;
    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    // Check if the descriptor was created successfully
    ASSERT_NE(matmulDesc, nullptr);

    // Set an attribute (Transpose A)
    hipblasOperation_t transa = HIPBLAS_OP_T;
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        matmulDesc,
        HIPBLASLT_MATMUL_DESC_TRANSA,   // Attribute to set
        &transa,                       // Host pointer
        sizeof(transa)                 // Attribute size
    ));

    // Clean up
    CUBLAS_CHECK(hipblasLtMatmulDescDestroy(matmulDesc));
}

TEST(cuBLASLt, MatrixLayoutCreateDestroy) {
    hipblasLtMatrixLayout_t layout;

    int64_t rows = 128;
    int64_t cols = 256;
    int64_t ld   = 128; // leading dimension

    // Create layout for a FP32 matrix
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layout, HIP_R_32F, rows, cols, ld));

    ASSERT_NE(layout, nullptr) << "Matrix layout should be successfully created";

    // Destroy layout
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layout));
}

TEST(cuBLASLt, MatmulPreferenceCreateSetAttributeDestroy) {
    hipblasLtMatmulPreference_t preference;

    // Create preference object
    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));

    ASSERT_NE(preference, nullptr) << "Matmul preference should be successfully created";

    // Set an attribute (for example, max workspace size)
    size_t maxWorkspaceSize = 1 << 20; // 1 MB
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(
        preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &maxWorkspaceSize,
        sizeof(maxWorkspaceSize)
    ));

    // Destroy preference object
    CUBLAS_CHECK(hipblasLtMatmulPreferenceDestroy(preference));
}

TEST(cuBLASLt, MatmulAlgoGetHeuristic) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Create MatmulDesc
    hipblasLtMatmulDesc_t matmulDesc;
    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    // Create Matrix Layouts (A, B, C)
    int64_t m = 128, n = 128, k = 128;
    hipblasLtMatrixLayout_t layoutA, layoutB, layoutC;
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layoutA, HIP_R_32F, m, k, m));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layoutB, HIP_R_32F, k, n, k));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layoutC, HIP_R_32F, m, n, m));

    // Create Matmul Preference
    hipblasLtMatmulPreference_t preference;
    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));

    size_t workspaceSize = 1 << 20; // 1 MB
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(
        preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &workspaceSize,
        sizeof(workspaceSize)
    ));

    // Query heuristics
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    int returnedResults = 0;

    CUBLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(
        (hipblasLtHandle_t) handle,
        matmulDesc,
        layoutA,
        layoutB,
        layoutC,
        layoutC, // D same as C
        preference,
        1, // Request just 1 heuristic
        &heuristicResult,
        &returnedResults
    ));

    // Validate we got at least one heuristic
    ASSERT_GT(returnedResults, 0) << "Expected at least one heuristic result";

    // Cleanup
    CUBLAS_CHECK(hipblasLtMatmulPreferenceDestroy(preference));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layoutA));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layoutB));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layoutC));
    CUBLAS_CHECK(hipblasLtMatmulDescDestroy(matmulDesc));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLASLt, MatmulBasic) {
    // --- Create a normal cuBLAS handle ---
    hipblasHandle_t cublasHandle;
    CUBLAS_CHECK(hipblasCreate(&cublasHandle));

    // --- Typecast to hipblasLtHandle_t for LtMatmul ---
    hipblasLtHandle_t ltHandle = reinterpret_cast<hipblasLtHandle_t>(cublasHandle);

    // --- Problem dimensions (very small GEMM) ---
    int m = 2, n = 2, k = 2;

    // Host data
    float h_A[4] = {1, 2, 3, 4};       // 2x2
    float h_B[4] = {5, 6, 7, 8};       // 2x2
    float h_C[4] = {0, 0, 0, 0};       // Output placeholder

    float alpha = 1.0f;
    float beta  = 0.0f;

    // --- Allocate device memory ---
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(h_A));
    hipMalloc(&d_B, sizeof(h_B));
    hipMalloc(&d_C, sizeof(h_C));

    hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeof(h_C), hipMemcpyHostToDevice);

    // --- Create descriptors ---
    hipblasLtMatmulDesc_t operationDesc;
    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    // Matrix layouts
    hipblasLtMatrixLayout_t layoutA, layoutB, layoutC;
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layoutA, HIP_R_32F, m, k, m));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layoutB, HIP_R_32F, k, n, k));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&layoutC, HIP_R_32F, m, n, m));

    // --- No preference (null means default algo heuristic) ---
    hipblasLtMatmulPreference_t preference;
    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));

    // Query heuristic (get a working algorithm)
    hipblasLtMatmulHeuristicResult_t heuristic;
    int returnedResults = 0;
    CUBLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(
        ltHandle,
        operationDesc,
        layoutA,
        layoutB,
        layoutC,
        layoutC,
        preference,
        1,
        &heuristic,
        &returnedResults
    ));
    ASSERT_GT(returnedResults, 0) << "No matmul heuristic found";

    // ✅ Check heuristic validity
    ASSERT_GE(heuristic.workspaceSize, 0) << "Workspace size invalid";

    // Optional: check if algoId looks reasonable (not always strictly necessary)
    ASSERT_NE(&heuristic.algo, nullptr) << "Invalid algorithm";

    // --- Run the matmul ---
    CUBLAS_CHECK(hipblasLtMatmul(
        ltHandle,
        operationDesc,
        &alpha,
        d_A, layoutA,
        d_B, layoutB,
        &beta,
        d_C, layoutC,
        d_C, layoutC,   // Output in same buffer as C
        &heuristic.algo,
        nullptr, 0,     // no workspace
        0               // stream 0
    ));

    // --- Copy back result ---
    hipMemcpy(h_C, d_C, sizeof(h_C), hipMemcpyDeviceToHost);

    // Expected GEMM result: C = A * B
    ASSERT_FLOAT_EQ(h_C[0], 23.0f);
    ASSERT_FLOAT_EQ(h_C[1], 34.0f);
    ASSERT_FLOAT_EQ(h_C[2], 31.0f);
    ASSERT_FLOAT_EQ(h_C[3], 46.0f);

    // --- Cleanup ---
    CUBLAS_CHECK(hipblasLtMatmulPreferenceDestroy(preference));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layoutA));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layoutB));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(layoutC));
    CUBLAS_CHECK(hipblasLtMatmulDescDestroy(operationDesc));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}
