/*
 * Written By: Theodoros Aslanidis <theodoros.aslanidis@ucdconnect.ie>
 *             School of Computer Science, University College Dublin
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <gtest/gtest.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA error: " << hipGetErrorString(err)
#define CUSPARSE_CHECK(err) \
    ASSERT_EQ((err), HIPSPARSE_STATUS_SUCCESS) << "CuSPARSE error: " << hipsparseGetErrorString(err)

TEST(CuSPARSE, CreateDestroy) {
    hipsparseHandle_t handle;
    CUSPARSE_CHECK(hipsparseCreate(&handle));
    CUSPARSE_CHECK(hipsparseDestroy(handle));
}

TEST(CuSPARSE, GetErrorString) {
    const char* msg = hipsparseGetErrorString(HIPSPARSE_STATUS_SUCCESS);
    ASSERT_NE(msg, nullptr);
}

TEST(CuSPARSE, GetVersion) {
    hipsparseHandle_t handle;
    CUSPARSE_CHECK(hipsparseCreate(&handle));
    int version;
    CUSPARSE_CHECK(hipsparseGetVersion(handle, &version));
    ASSERT_GT(version, 999);  // Version should be greater than 1.0
    CUSPARSE_CHECK(hipsparseDestroy(handle));
}

TEST(CuSPARSE, SetGetStream) {
    hipsparseHandle_t handle;
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseSetStream(handle, stream));

    hipStream_t returned_stream;
    CUSPARSE_CHECK(hipsparseGetStream(handle, &returned_stream));
    ASSERT_EQ(stream, returned_stream);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUSPARSE_CHECK(hipsparseDestroy(handle));
}