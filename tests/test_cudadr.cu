/*
 * Written By: Theodoros Aslanidis <theodoros.aslanidis@ucdconnect.ie>
 *             School of Computer Science, University College Dublin
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)

TEST(cudaDR, getDriverVersion) {
    int version = 0;
    CUDA_CHECK(hipDriverGetVersion(&version));
    ASSERT_GT(version, 0);
}