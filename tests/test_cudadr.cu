#include <gtest/gtest.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)

TEST(cudaDR, getDriverVersion) {
    int version = 0;
    CUDA_CHECK(hipDriverGetVersion(&version));
    ASSERT_GT(version, 0);
}