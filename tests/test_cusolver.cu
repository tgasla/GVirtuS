/*
 * Written By: Theodoros Aslanidis <theodoros.aslanidis@ucdconnect.ie>
 *             School of Computer Science, University College Dublin
 */

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <gtest/gtest.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
#define CUSOLVER_CHECK(err) ASSERT_EQ((err), HIPSOLVER_STATUS_SUCCESS)

TEST(CuSolver, CreateDestroy) {
    hipsolverHandle_t handle;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
}

TEST(CuSolver, GetSetStream) {
    hipsolverHandle_t handle;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSOLVER_CHECK(hipsolverSetStream(handle, stream));

    hipStream_t returned_stream;
    CUSOLVER_CHECK(hipsolverGetStream(handle, &returned_stream));
    ASSERT_EQ(stream, returned_stream);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
}